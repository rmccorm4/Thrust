
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
	int runtime;
	int driver;

	hipError_t runtime_err = hipRuntimeGetVersion(&runtime);
	hipError_t driver_err = hipRuntimeGetVersion(&driver);

	printf("Runtime Version: %d. %s\n", runtime, hipGetErrorString(runtime_err));
	printf("Driver Version: %d. %s\n", driver, hipGetErrorString(driver_err));
}

